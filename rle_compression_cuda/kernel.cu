#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <chrono>
#include <filesystem>
#include <vector>
#include <fstream>
#include <iterator>

#include "hip/hip_runtime.h"
#include ""

using namespace std;

int THREADS_PER_BLOCK = 512;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

__global__ void generateMask(uint8_t *input, uint32_t *mask, uint64_t blockCount)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (index == 0)
    {
        mask[0] = 1;
        for (int i = index + 1; i < blockCount; i += stride)
        {
            mask[i] = input[i] == input[i - 1] ? 0 : 1;
        }
        return;
    }

    for (int i = index; i < blockCount - 1; i += stride)
    {
        mask[i] = input[i] == input[i - 1] ? 0 : 1;
    }
}

void sequentialScan(uint32_t *output, uint32_t *input, uint64_t blockCount, bool inclusive)
{
    if (inclusive) {
        output[0] = input[0];
        for (int j = 1; j < blockCount; ++j)
        {
            output[j] = input[j] + output[j - 1];
        }
    }
    else {
        output[0] = 0;
        for (int j = 1; j < blockCount; ++j)
        {
            output[j] = input[j - 1] + output[j - 1];
        }
    }
}

__global__ void scan(uint32_t *output, uint32_t *input, uint32_t *sums, uint64_t n, bool inclusive)
{
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * n;

    extern __shared__ int temp[];
    temp[2 * threadID] = input[blockOffset + (2 * threadID)];
    temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

    int offset = 1;
    for (int d = n >> 1; d > 0; d >>= 1)
    {
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    __syncthreads();


    if (threadID == 0) {
        sums[blockID] = temp[n - 1];
        temp[n - 1] = 0;
    }

    for (int d = 1; d < n; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    if (inclusive)
    {
        output[blockOffset + (2 * threadID) - 1] = temp[2 * threadID];
        output[blockOffset + (2 * threadID)] = temp[2 * threadID + 1];
    }
    else
    {
        output[blockOffset + (2 * threadID)] = temp[2 * threadID];
        output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
    }
} 

__global__ void inclusiveAdd(uint32_t* output, uint32_t length, uint32_t* n) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * length - 1;

    output[blockOffset + threadID] += n[blockID - 1];
}

__global__ void add(uint32_t* output, uint32_t length, uint32_t* n) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * length;

    output[blockOffset + threadID] += n[blockID];
}

__global__ void compact(uint32_t* scannedMask, uint32_t* compactedMask, uint32_t* totalSize, uint64_t n) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (index == 0) {
        compactedMask[0] = 0;
    }

    for (int i = index; i < n; i += stride) {

        if (i == (n - 1)) {
            compactedMask[scannedMask[i]] = i + 1;
            *totalSize = scannedMask[i];
        }

        if (scannedMask[i] != scannedMask[i - 1]) {
            compactedMask[scannedMask[i] - 1] = i;
        }
    }
}

__global__ void scatter(uint32_t* compactedMask, uint32_t* totalSize, uint8_t* input, uint8_t* outputData, uint32_t* occurences) {

    int n = *totalSize;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        int a = compactedMask[i];
        int b = compactedMask[i + 1];

        outputData[i] = input[a];
        occurences[i] = b - a;
    }
}

void prefixSum(const int gridSize, uint64_t blockCount, uint32_t* mask, uint32_t* scannedMask, bool inclusive)
{
    const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

    uint32_t* blockSums;
    uint32_t* scannedBlockSums;

    hipMallocManaged(&blockSums, blockCount * sizeof(uint32_t));
    hipMallocManaged(&scannedBlockSums, gridSize * sizeof(uint32_t));

    scan<<<gridSize, THREADS_PER_BLOCK, sharedMemArraySize >> > (scannedMask, mask, blockSums, ELEMENTS_PER_BLOCK, inclusive);
    hipDeviceSynchronize();

    const int smallGridSize = (gridSize + ELEMENTS_PER_BLOCK - 1) / ELEMENTS_PER_BLOCK;

    // if scan size is smaller than viable for parallel scan do it sequentialy
    if (smallGridSize < 2) {
        sequentialScan(scannedBlockSums, blockSums, gridSize, inclusive);
    }
    else
    {
        prefixSum(smallGridSize, gridSize, blockSums, scannedBlockSums, inclusive);
    }

    if (inclusive)
    {
        inclusiveAdd<<<gridSize, ELEMENTS_PER_BLOCK>>>(scannedMask, ELEMENTS_PER_BLOCK, scannedBlockSums);
    }
    else
    {
        add<<<gridSize, ELEMENTS_PER_BLOCK>>>(scannedMask, ELEMENTS_PER_BLOCK, scannedBlockSums);
    }
    hipDeviceSynchronize();

    hipFree(blockSums);
    hipFree(scannedBlockSums);
}

void compress(const string filename)
{
    ifstream inputFile;

    const uint64_t filesize = filesystem::file_size(filesystem::path(filename));
    const uint64_t blockCount = filesize / sizeof(uint8_t);
    const int gridSize = (blockCount + ELEMENTS_PER_BLOCK - 1) / ELEMENTS_PER_BLOCK;
    uint32_t *scannedMask;
    uint32_t *compactedMask;
    uint32_t *sequentialScannedMask = new uint32_t[blockCount];
    uint32_t *mask;
    uint32_t *totalSize;
    uint32_t *occurences;
    uint8_t *outputData;
    uint8_t *memblock;

    hipMallocManaged(&memblock, blockCount * sizeof(uint8_t));
    hipMallocManaged(&mask, blockCount * sizeof(uint32_t));
    hipMallocManaged(&scannedMask, blockCount * sizeof(uint32_t));
    hipMallocManaged(&totalSize, blockCount * sizeof(uint32_t));
    hipMallocManaged(&compactedMask, blockCount * sizeof(uint32_t));

    inputFile.open(filename, ios::binary);
    inputFile.read((char *)memblock, blockCount * sizeof(uint8_t));

    auto t1 = chrono::high_resolution_clock::now();

    generateMask<<<gridSize, THREADS_PER_BLOCK>>>(memblock, mask, blockCount);
    hipDeviceSynchronize();

    prefixSum(gridSize, blockCount, mask, scannedMask, true);

    compact<<<gridSize, THREADS_PER_BLOCK>>>(scannedMask, compactedMask, totalSize, blockCount);
    hipDeviceSynchronize();

    hipMallocManaged(&outputData, *totalSize * sizeof(uint8_t));
    hipMallocManaged(&occurences, *totalSize * sizeof(uint32_t));

    scatter<<<gridSize, THREADS_PER_BLOCK>>>(compactedMask, totalSize, memblock, outputData, occurences);
    hipDeviceSynchronize();

    auto t2 = chrono::high_resolution_clock::now();
    auto ms_int = chrono::duration_cast<chrono::milliseconds>(t2 - t1);
    cout << filename << " GPU compression time: " << ms_int.count() << "ms\n";
    
    ofstream outputFile;
    outputFile.open(filename + ".rlz", ios::binary);
    outputFile.write((char*) totalSize, sizeof(uint32_t));
    outputFile.write((char*) outputData, *totalSize * sizeof(uint8_t));
    outputFile.write((char*) occurences, *totalSize * sizeof(uint32_t));

    hipFree(memblock);
    hipFree(mask);
    hipFree(scannedMask);
    hipFree(compactedMask);
    hipFree(totalSize);
    hipFree(outputData);
    hipFree(occurences);
}

__global__ void generateDecompressedData(uint8_t* input, uint8_t* output, uint32_t* occurences, uint32_t* positions, uint64_t blockCount)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < blockCount; i += stride)
    {
        int position = positions[i];
        int occurence = occurences[i];

        for (int j = position; j < position + occurence; j++)
        {
            output[j] = input[i];
        }
    }
}

void decompress(const string filename)
{
    ifstream inputFile;

    inputFile.open(filename, ios::binary);

    uint32_t totalSize;
    uint32_t* occurences;
    uint32_t* scannedOccurences;
    uint8_t* compressedData;
    uint8_t* decompressedData;
    uint64_t decompressedSize;


    inputFile.read((char*)&totalSize, sizeof(uint32_t));

    hipMallocManaged(&occurences, totalSize * sizeof(uint32_t));
    hipMallocManaged(&compressedData, totalSize * sizeof(uint8_t));
    hipMallocManaged(&scannedOccurences, totalSize * sizeof(uint32_t));

    inputFile.read((char*)compressedData, totalSize * sizeof(uint8_t));
    inputFile.read((char*)occurences, totalSize * sizeof(uint32_t));

    const int gridSize = (totalSize + ELEMENTS_PER_BLOCK - 1) / ELEMENTS_PER_BLOCK;

    auto t1 = chrono::high_resolution_clock::now();

    prefixSum(gridSize, totalSize, occurences, scannedOccurences, false);
    hipDeviceSynchronize();

    decompressedSize = scannedOccurences[totalSize - 1] + occurences[totalSize - 1];
    hipMallocManaged(&decompressedData, decompressedSize * sizeof(uint8_t));

    const int bigGridSize = (totalSize + THREADS_PER_BLOCK + 1) / THREADS_PER_BLOCK;
    generateDecompressedData<<<bigGridSize, THREADS_PER_BLOCK>>>(compressedData, decompressedData, occurences, scannedOccurences, totalSize);
    hipDeviceSynchronize();

    auto t2 = chrono::high_resolution_clock::now();
    auto ms_int = chrono::duration_cast<chrono::milliseconds>(t2 - t1);
    cout << filename << " GPU decompression time: " << ms_int.count() << "ms\n" << endl;

    ofstream outputFile;
    outputFile.open(filename + "_decompressed.bmp", ios::binary);
    outputFile.write((char*)decompressedData, decompressedSize * sizeof(uint8_t));
}


int main(int argc, char const *argv[])
{
    string filename = "image.bmp";
    compress(filename);
    decompress(filename + ".rlz");

    filename = "simple_image.bmp";
    compress(filename);
    decompress(filename + ".rlz");
}
