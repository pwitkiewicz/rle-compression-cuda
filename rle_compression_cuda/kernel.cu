﻿#include <iostream>
#include <chrono>
#include <filesystem>
#include <vector>
#include <fstream>

#include "hip/hip_runtime.h"


using namespace std;

__global__ void backwardMask(uint8_t *input, uint8_t *mask, uint64_t blockCount)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (index == 0)
    {
        mask[0] = 1;
        for (int i = index + 1; i < blockCount; i += stride)
        {
            mask[i] = input[i] == input[i - 1] ? 0 : 1;
        }
        return;
    }

    for (int i = index; i < blockCount; i += stride)
    {
        mask[i] = input[i] == input[i - 1] ? 0 : 1;
    }
}

void compress(const string filename)
{
    ifstream inputFile;

    uint64_t filesize = filesystem::file_size(filesystem::path(filename));
    uint64_t blockCount = filesize / sizeof(uint8_t);
    uint32_t gridSize = (blockCount + 512 - 1) / 512;
    uint8_t *memblock;
    uint8_t* mask;
    

    hipMallocManaged(&memblock, blockCount * sizeof(uint8_t));
    hipMallocManaged(&mask, blockCount * sizeof(uint8_t));

    inputFile.open(filename, ios::binary);
    inputFile.read((char *)memblock, blockCount * sizeof(uint8_t));

    backwardMask<<<gridSize, 512>>>(memblock, mask, blockCount);
    hipDeviceSynchronize();

    hipFree(memblock);
    hipFree(mask);
    //runLengthEncode(memblock, outputData, counter, blockCount);
    //writeCompressedFile(filename, outputData, counter);
}

int main(int argc, char const *argv[])
{
    string filename = "simple_image.bmp";

    auto t1 = chrono::high_resolution_clock::now();
    compress(filename);
    auto t2 = chrono::high_resolution_clock::now();
    auto ms_int = chrono::duration_cast<chrono::milliseconds>(t2 - t1);
    cout << filename << " GPU compression time: " << ms_int.count() << "ms\n";
}
